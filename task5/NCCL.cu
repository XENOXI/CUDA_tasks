#include "hip/hip_runtime.h"
#include <iostream>
#include <mpi.h>
#include <cmath>
#include <string>
#include <sstream>
#include <exception>
#include <cstring>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define CREATE_DEVICE_ARR(type,arg,size) type* arg; hipMalloc((void**)&arg, sizeof(type) * size);
#define CUDACHECK(name) if (hipGetLastError() != hipSuccess || hipDeviceSynchronize() != hipSuccess) throw std::runtime_error(name);
#define NCCLCHECK(cmd) do {                               \
    ncclResult_t r = cmd;                             \
    if (r!= ncclSuccess) {                            \
      printf("Failed, NCCL error %s:%d '%s'\n",       \
          __FILE__,__LINE__,ncclGetErrorString(r));   \
      exit(EXIT_FAILURE);                             \
    }}                                                \
    while(0)
  

template <typename Type>
std::istream& operator>>(std::istream& i, const Type& arg) { return i; }

template <typename Type>
void argpars(Type& arg, std::string& str)
{
	std::stringstream buff;
	buff << str;
	buff >> arg;
	std::string buff2;
	buff2 = str;
	str.clear();
	std::getline(buff, str);
	if (str == buff2) //Nothing changed
		throw std::runtime_error("Not a valid argument");
}

__global__ void interpolate(double* A,double* Anew,unsigned int size_x,unsigned int size_y)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x==0 || y==0 || x==size_x-1 || y==size_y-1)
        return;
    Anew[y*size_x+x] = (A[y*size_x+x-1] + A[y*size_x+x+1] + A[(y-1)*size_x+x] + A[(y+1)*size_x+x]) / 4;
}

__global__ void difference(double* A,double* B)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    A[i] -= B[i];
}

int NOD(int a, int b)
{
    while(a > 0 && b > 0)
        if(a > b)
            a %= b;
        else
            b %= a;
    return a + b;
}

int main(int argc,char *argv[])
{
    //MPI init   
    int rank,threads_cnt;
    MPI_Init(&argc,&argv);
    
    MPI_Comm_rank(MPI_COMM_WORLD,&rank);
    MPI_Comm_size(MPI_COMM_WORLD,&threads_cnt);

    //CUDA check device count
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount<threads_cnt)
        throw std::runtime_error("Too many MPI threads");
    hipSetDevice(rank);

    //Stream init
    hipStream_t s;
    hipStreamCreate(&s);

    //Set p2p access
    if (rank!=0)
        hipDeviceEnablePeerAccess(rank-1,0);
    if (rank!=threads_cnt-1)
        hipDeviceEnablePeerAccess(rank+1,0);

    //default settings
    double accuracy = std::pow(10,-6);
    unsigned int net_len=1024;
    unsigned int iteration_cnt = std::pow(10,6);
    
    //NCCL init 
    ncclUniqueId id;
    ncclComm_t comm;
    if (rank == 0) ncclGetUniqueId(&id);
    MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
    ncclCommInitRank(&comm, threads_cnt, id, rank);
    

    //Reading arguments
    for (int i =1;i<argc-1;i+=2)
    {
        std::string argument(argv[i]);
        std::string value(argv[i+1]);

        if (argument=="-accuracy")
            argpars(accuracy,value);
        else if (argument=="-net_len")
            argpars(net_len,value);
        else if (argument=="-iteration")
            argpars(iteration_cnt,value);
    }

    //Init net and buffer

    //Matrix is divided so that each process has about net_len/threads_cnt rows +
    // 2 rows if rank not first or last
    // 1 row if first or last
    unsigned int start = net_len*rank/threads_cnt-1;
    unsigned int end = net_len*(rank+1)/threads_cnt+1;

    if (rank==0)
        start+=1;
    if (rank==threads_cnt-1)
        end-=1;

    unsigned int net_len_per_gpu = end-start;
    if (threads_cnt==1)
        net_len_per_gpu=net_len;
    
    unsigned int net_size = net_len_per_gpu*net_len;
    
    double* net_cpu = new double[net_size];
    memset(net_cpu,0,net_size*sizeof(double));

    CREATE_DEVICE_ARR(double,buff,net_size)
    CREATE_DEVICE_ARR(double,net,net_size)
    CREATE_DEVICE_ARR(double,net_buff,net_size)
    CREATE_DEVICE_ARR(char,is_end,1)
    CREATE_DEVICE_ARR(double,d_out,1)
    CREATE_DEVICE_ARR(double,d_in,1)
    
    //Corners
    double lu = 10;
    double ru = 20;
    double ld = 30;
    double rd = 20;

    //Threads and blocks init
    unsigned int threads_x=NOD(net_len,1024);
    unsigned int blocks_y = net_len_per_gpu;
    unsigned int blocks_x = net_len/threads_x;

    dim3 dim_for_interpolate(threads_x,1);
    dim3 block_for_interpolate(blocks_x,blocks_y);

    //Fill default values
    if (rank==0)
        for (int i =0;i<net_len;i++)
            net_cpu[i] = (ru-lu)/(net_len-1)*i + lu;
        
    if (rank==threads_cnt-1)
        for (int i =0;i<net_len;i++)
            net_cpu[i+(net_len_per_gpu-1)*net_len] = (rd-ld)/(net_len-1)*i + ld;

    for (int i =0;i<net_len_per_gpu;i++)
    {
        net_cpu[net_len*i] = (ld-lu)/(net_len-1)*(i+start) + lu;
        net_cpu[net_len-1 + net_len*i] = (rd-ru)/(net_len-1)*(i+start) + ru;
    }

    hipMemcpy(net,net_cpu, sizeof(double)*net_size, hipMemcpyHostToDevice);
    hipMemcpy(net_buff,net_cpu, sizeof(double)*net_size, hipMemcpyHostToDevice);

    //Cub init
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, buff, d_out, net_size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);


    //Init cycle values
    unsigned int iter;
    double max_acc=0;

    //Start solving
    for (iter = 0;iter <iteration_cnt;iter++)
    {  
        //Set the new array
        interpolate<<<block_for_interpolate,dim_for_interpolate,0,s>>>(net,net_buff,net_len,net_len_per_gpu);  

        //Doing reduction to find max accuracy
        if (iter % 100 == 0 || iter == iteration_cnt-1)
        {
            hipMemcpy(buff,net_buff, sizeof(double)*net_size, hipMemcpyDeviceToDevice);
            difference<<<blocks_x*blocks_y,threads_x,0,s>>>(buff,net);
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, buff, d_out, net_size,s);
            
            //Sending max accuracy to all GPU
            NCCLCHECK(ncclGroupStart());
            NCCLCHECK(ncclAllReduce((void*)d_out,(void*)d_in,1,ncclDouble,ncclMax,comm,s));
            NCCLCHECK(ncclGroupEnd());

            //Sending accuracy to host
            hipMemcpyAsync(&max_acc,d_in,sizeof(double),hipMemcpyDeviceToHost,s);
            hipStreamSynchronize(s);
            if(max_acc<accuracy)
                break;           
        }

        //Exchanging matrix rows between GPUs
        //This send penultimate and second rows 
        //and get last and fisrt rows 
        NCCLCHECK(ncclGroupStart());
        if (rank!=threads_cnt-1)
        {
            
            NCCLCHECK(ncclSend(&net_buff[(net_len_per_gpu-2)*net_len+1],net_len-2,ncclDouble,rank+1,comm,s));
            NCCLCHECK(ncclRecv(&net_buff[(net_len_per_gpu-1)*net_len+1],net_len-2,ncclDouble,rank+1,comm,s));
        }
        if (rank!=0)
        {
            NCCLCHECK(ncclSend(&net_buff[net_len+1],net_len-2,ncclDouble,rank-1,comm,s)); 
            NCCLCHECK(ncclRecv(&net_buff[1],net_len-2,ncclDouble,rank-1,comm,s));
        }
        
        NCCLCHECK(ncclGroupEnd());
        std::swap(net,net_buff);
              
    }
    CUDACHECK("end")
    

    //Printing results
    if (rank==0)
    {
        std::cout<<"Iteration count: "<<iter<<"\n";
        std::cout<<"Accuracy: "<<max_acc<<"\n";
    }


    //Finishing program
    hipFree(net);
    hipFree(net_buff);
    hipFree(buff);
    hipFree(d_out);
    delete[] net_cpu;
    ncclCommDestroy(comm);
    MPI_Finalize();
    return 0;
}


