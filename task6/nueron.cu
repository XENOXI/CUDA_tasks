#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define CREATE_DEVICE_ARR(type,arg,size) type* arg; hipMalloc((void**)&arg, sizeof(type) * size);

typedef struct farray
{
    float* data;
    uint32_t size;
}farray;

template <class T>
std::vector<T> get_array(std::string filepath)
{
    std::ifstream file(filepath);
    std::vector<T> arr;
    T buff;
    while (!file.eof())
    {
        file >> buff;
        arr.push_back(buff);
    }
    arr.pop_back();
    return arr;
}

uint32_t NOD(uint32_t a, uint32_t b)
{
    while(a > 0 && b > 0)
        if(a > b)
            a %= b;
        else
            b %= a;
    return a + b;
}

std::vector<float> transpose(std::vector<float> in,uint32_t a,uint32_t b)
{
    std::vector<float> out(a*b);
    for(uint32_t i = 0; i < a; i++)
        for(uint32_t j = i; j < b; j++)
            out[j*a+i] = in[i*b+j];
    return out;
}

class Layer
{
protected:
    farray in_x;
    farray out_x;
    farray grad;
    farray err_x;
    dim3 threads={1,1,1};
    dim3 blocks={1,1,1};
public:
    virtual farray forward(farray x) = 0;
    virtual farray backward(farray err)=0;
    virtual void read_weights(std::string filepath) {};
};




class Linear : public Layer
{
private:
    farray weights;
    farray buff;
    hipblasHandle_t handle;
    uint32_t in_size;
    uint32_t out_size;
    float alpha = 1;
    float beta = 0;
public:
    Linear(uint32_t in, uint32_t out) : in_size(in), out_size(out)
    {
        hipMalloc((void**)&weights.data,in*out*sizeof(float));
        hipMalloc((void**)&grad.data,in*out*sizeof(float));
        hipMalloc((void**)&err_x.data,in*sizeof(float));
        weights.size = in*out;
        hipblasCreate(&handle);
    }
    farray forward(farray x)
    {
        in_x = x;
        hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,1,out_size,in_size,&alpha,x.data,1,weights.data,in_size,&beta,out_x.data,1);
        return out_x;
    }
    farray backward(farray err)
    {
        hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,1,out_size,in_size,&alpha,err.data,1,weights.data,in_size,&beta,err_x.data,1);
        hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,in_size,out_size,1,&alpha,in_x.data,in_size,err.data,1,&beta,grad.data,in_size);
        return err_x;
    }
    void read_weights(std::string filepath)
    {
        hipMemcpy(weights.data,transpose(get_array<float>(filepath),out_size,in_size).data(),weights.size*sizeof(float),hipMemcpyHostToDevice);
    }

};

class Sigmoid : public Layer
{
private:
    __global__ void sigm_forward(float* in,float* out)
    {
        uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
        out[id] = 1/(1+exp(-in[id]));
    }
    __global__ void sigm_backward(float* err,float* out,float* grad)
    {
        uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
        grad[id] = err[id]*out[id]*(1-out[id]);
    }
public:
    Sigmoid(uint32_t size) {
        hipMalloc((void**)&out_x.data,size*sizeof(float));
        hipMalloc((void**)&err_x.data,size*sizeof(float));
        threads.x = NOD(size,1024);
        blocks.x = 1024/threads.x;
        out_x.size = size;
        in_x.size = size;
    }
    farray forward(farray x)
    {
        in_x = x;
        sigm_forward<<<blocks,threads>>>(x.data,out_x.data);
        return out_x;
    }
    farray backward(farray err)
    {
        sigm_backward<<<blocks,threads>>>(err.data,out_x.data,err_x.data);
        return err_x;
    }
    void read_weights(std::string filepath) {}
};

class Model
{
public:
    std::vector<Layer*> layers;
    farray forward(farray x)
    {
        for (auto lay : layers)
            x = lay->forward(x);
        return x;
    }
    void backward(farray out)
    {
        for (auto lay : layers)
            out = lay->backward(out);
    }
   
};

int main()
{
    auto vec = get_array<float>("weight.txt");

    auto net = Model();
    net.layers.push_back(new Linear(32 * 32, 16 * 16));
    net.layers.push_back(new Sigmoid(16*16));
    net.layers.push_back(new Linear(16 * 16,4*4));
    net.layers.push_back(new Sigmoid(4*4));
    net.layers.push_back(new Linear(4*4,1));
    net.layers.push_back(new Sigmoid(1));


    farray in;
    hipMalloc((void**)&in.data,sizeof(float)*32*32);
    hipMemset(in.data,0,sizeof(float)*32*32);
    
    
    std::cout << net.forward(in).data[0] << std::endl;
    return 0;
}